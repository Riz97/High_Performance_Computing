#include "hip/hip_runtime.h"
#include <stdio.h>


//Number of blocks , number of threads per block
//<<<1,1>>>
//<<<1,10>>>
//<<<10,1>>>
//<<<10,10>>>




__global__ void GPUFunction()
{
    printf("This is running in parallel.\n");
}

int main()
{
    GPUFunction <<<5,5>>>();
    cudaDeviceSynchronization();
}