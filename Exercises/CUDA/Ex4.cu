#include "hip/hip_runtime.h"
#include <stdio.h>

//2D grid

__global__ void matrixAdd(float a[N][N],float b[N][N],float c[N][N])
{
    //every thread compute an element
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j =  blockIdx.y * blockDim.y + threadIdx.y;

    c[i][j] = a[i][j] + b[i][j]
    
}

int main(){

    dim3 blocksPerGrid(N/16,N/16); // 1 block per grid  , only one block
    dim3 threadsPerBlock(16,16);// NxN threads per blocks , inside this block we have NxN threads
    matrixAdd<<<blocksPerGrid,threadsPerBlock>>>(a,b,c);
}