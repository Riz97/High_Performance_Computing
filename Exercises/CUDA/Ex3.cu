#include "hip/hip_runtime.h"
#include <stdio.h>

//1D grid

__global__ void matrixAdd(float a[N][N],float b[N][N],float c[N][N])
{
    //every thread compute an element
    int i = threadIdx.x;
    int j = threadIdx.y;

    c[i][j] = a[i][j] + b[i][j]
    
}

int main(){

    dim3 blocksPerGrid(1); // 1 block per grid  , only one block
    dim3 threadsPerBlock(N,N);// NxN threads per blocks , inside this block we have NxN threads
    matrixAdd<<<blocksPerGrid,threadsPerBlock>>>(a,b,c);
}