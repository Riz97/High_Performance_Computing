
#include <hip/hip_runtime.h>
#include <stdio.h>

void CPUFunction()
{
    printf("Hello World from the CPU\n");
}

__global__ void GPUFunction()
{
    printf("Hello World from the GPU \n");

}


int main(){
    CPUFunction();
    GPUFunction<<<1,1>>>();
    hipDeviceSynchronize();
}